#include "hip/hip_runtime.h"
#include "sgemm.cuh"
#include <iostream>
#include <hipblas.h>
#include <functional>
#include <random>
#include <string>
#include <cstdio>
using namespace std;

// MARCO
#define OFFSET(i, j, N) (i) * (N) + (j)
#define FLOAT4(pointer) reinterpret_cast<float4*>(&(pointer))[0]

#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t err = call;                                                \
        if (err != hipSuccess) {                                              \
            printf("CUDA Error: \n");                                          \
            printf("    File:       %s\n", __FILE__);                          \
            printf("    Line:       %d\n", __LINE__);                          \
            printf("    Error Code: %d\n", err);                               \
            printf("    Error Text: %s\n", hipGetErrorString(err));           \
            exit(1);                                                           \
        }                                                                      \
    } while (0)

#define CUBLAS_CHECK(call)                                                     \
    do {                                                                       \
        hipblasStatus_t err = call;                                             \
        if (err != HIPBLAS_STATUS_SUCCESS) {                                    \
            printf("cuBLAS Error: \n");                                        \
            printf("    File:       %s\n", __FILE__);                          \
            printf("    Line:       %d\n", __LINE__);                          \
            printf("    Error Code: %d\n", err);                               \
            printf("    Error Text: %s\n", cublasGetStatusString(err));        \
            exit(1);                                                           \
        }                                                                      \
    } while (0)


// Naive
__global__ void sgemm_v1_kernel(float *a, float *b, float*c, int M, int N, int K){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for(int k = 0; k < K; k++)
        c[OFFSET(row, col, N)] += a[OFFSET(row, k, K)] * b[OFFSET(k, col, N)];
}

float sgemm_v1(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;

    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 16, BLOCK_N = 16;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N, BLOCK_M);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v1_kernel<<<grid, block>>>(da, db, dc, M, N, K);

    hipGetLastError();
    hipDeviceSynchronize();    
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecond, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return msecond;
}



// Block Tiling
__global__ void sgemm_v2_kernel(float *a, float *b, float *c, int M, int N, int K){

    const int BLOCK_M = 16, BLOCK_N = 16, BLOCK_K = 64;

    __shared__ float shared_a[BLOCK_M][BLOCK_K];
    __shared__ float shared_b[BLOCK_K][BLOCK_N];

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 16;
    int smem_a_k = (tid % 16) << 2;

    int smem_b_k = tid / 4;
    int smem_b_n = (tid % 4) << 2;

    int gmem_a_m = smem_a_m + blockIdx.y * BLOCK_M;
    int gmem_b_n = smem_b_n + blockIdx.x * BLOCK_N;

    float result = 0.0;
    for(int k = 0; k < K / BLOCK_K; k++){
        // GMEM copy to SMEM
        int gmem_a_k = smem_a_k + k * BLOCK_K;
        int gmem_b_k = smem_b_k + k * BLOCK_K;

        FLOAT4(shared_a[smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        FLOAT4(shared_b[smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);

        __syncthreads();

        // Compute
        for(int kk = 0; kk < BLOCK_K; kk++)
            result += shared_a[threadIdx.y][kk] * shared_b[kk][threadIdx.x];
        
        __syncthreads();
    } 

    // Write back
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[OFFSET(row, col, N)] = result;
}

float sgemm_v2(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;


    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 16, BLOCK_N = 16;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N, BLOCK_M);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v2_kernel<<<grid, block>>>(da, db, dc, M, N, K);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecond, start, stop);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return msecond;
}

// Thread Tiling
__global__ void sgemm_v3_kernel(float *a, float *b, float*c, int M, int N, int K){
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 8;
    const int TILE = 8;

    __shared__ float shared_a[BLOCK_M][BLOCK_K];
    __shared__ float shared_b[BLOCK_K][BLOCK_N];

    float reg_a[TILE];
    float reg_b[TILE];
    float reg_c[TILE][TILE];


    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) << 2;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) << 2;

    int gmem_a_m = smem_a_m + blockIdx.y * BLOCK_M;
    int gmem_b_n = smem_b_n + blockIdx.x * BLOCK_N;

    for(int k = 0; k < K / BLOCK_K; k++){
        int gmem_a_k = smem_a_k + k * BLOCK_K;
        int gmem_b_k = smem_b_k + k * BLOCK_K;

        // Copy
        FLOAT4(shared_a[smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        FLOAT4(shared_b[smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
        __syncthreads();

        // Compute
        int ty = threadIdx.y * TILE;
        int tx = threadIdx.x * TILE;

        for(int kk = 0; kk < BLOCK_K; kk++){
            for(int i = 0; i < TILE; i++) {
                reg_a[i] = shared_a[ty + i][kk];
                reg_b[i] = shared_b[kk][tx + i];
            }
            for(int i = 0; i < TILE; i++)
                for(int j = 0; j < TILE; j++)
                    reg_c[i][j] += reg_a[i] * reg_b[j];
        }
        __syncthreads();
    }

    // Write Back
    int row = blockIdx.y * BLOCK_M + threadIdx.y * TILE;
    int col = blockIdx.x * BLOCK_N + threadIdx.x * TILE;

    for(int i = 0; i < TILE; i++)
        for(int j = 0; j < TILE; j += 4)
            FLOAT4(c[OFFSET(row + i, col + j, N)]) = FLOAT4(reg_c[i][j]); 
}
float sgemm_v3(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;


    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 128, BLOCK_N = 128, TILE = 8;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N / TILE, BLOCK_M / TILE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v3_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecond, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return msecond;
}

// Warp Tiling
__global__ void sgemm_v4_kernel(float *a, float *b, float*c, int M, int N, int K){
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 8;
    const int TILE = 8, NUM = 4;

    __shared__ float shared_a[BLOCK_M][BLOCK_K];
    __shared__ float shared_b[BLOCK_K][BLOCK_N];

    float reg_a[TILE];
    float reg_b[TILE];
    float reg_c[TILE][TILE] = {0};

    int tid = threadIdx.y * blockDim.x + threadIdx.x;


    int warp_id = tid / 32;
    int lane_id = tid % 32;
    
    int CTA_Layout[2] = {4, 2};
    int WARP_Layout[2] = {4, 8};

    int WARP_M = BLOCK_M / CTA_Layout[0];
    int WARP_N = BLOCK_N / CTA_Layout[1];

    int warp_y = warp_id / CTA_Layout[1];
    int warp_x = warp_id % CTA_Layout[1];

    int lane_y = lane_id / WARP_Layout[1];
    int lane_x = lane_id % WARP_Layout[1];

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) << 2;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) << 2;

    int gmem_a_m = smem_a_m + blockIdx.y * BLOCK_M;
    int gmem_b_n = smem_b_n + blockIdx.x * BLOCK_N;

    for(int k = 0; k < K / BLOCK_K; k++){
        int gmem_a_k = smem_a_k + k * BLOCK_K;
        int gmem_b_k = smem_b_k + k * BLOCK_K;

        // Copy
        FLOAT4(shared_a[smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        FLOAT4(shared_b[smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
        __syncthreads();

        // Compute
        int wy = warp_y * WARP_M;
        int wx = warp_x * WARP_N;
        
        for(int kk = 0; kk < BLOCK_K; kk++){
            int ty = wy + lane_y * TILE;
            int tx = wx + lane_x * NUM;

            for(int i = 0; i < TILE; i++) 
                reg_a[i] = shared_a[ty + i][kk];
            
            for(int per = 0; per < 2; per++)
                for(int i = 0; i < NUM; i++)
                    reg_b[i + per * NUM] = shared_b[kk][tx + i + per * WARP_N / 2];
            
            // Compute NUM = 4
            for(int i = 0; i < TILE; i++)
                for(int j = 0; j < TILE; j++)
                    reg_c[i][j] += reg_a[i] * reg_b[j];
        }
        __syncthreads();
    }

    // Write Back
    int row = blockIdx.y * BLOCK_M + warp_y * WARP_M + lane_y * TILE;
    int col = blockIdx.x * BLOCK_N + warp_x * WARP_N + lane_x * NUM;


    for(int m = 0; m < TILE; m++){
        int r_c_1 = row + m;
        int c_c_1 = col;
        int c_c_2 = c_c_1 + WARP_N / 2;
        FLOAT4(c[OFFSET(r_c_1, c_c_1, N)]) = FLOAT4(reg_c[m][0]);
        FLOAT4(c[OFFSET(r_c_1, c_c_2, N)]) = FLOAT4(reg_c[m][NUM]);
    }
}

float sgemm_v4(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;


    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 128, BLOCK_N = 128, TILE = 8;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N / TILE, BLOCK_M / TILE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v4_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecond, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return msecond;
}

// Bank Free
__global__ void sgemm_v5_kernel(float *a, float *b, float*c, int M, int N, int K){
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 8;
    const int TILE = 8, NUM = 4;

    __shared__ float shared_a[BLOCK_M][BLOCK_K];
    __shared__ float shared_b[BLOCK_K][BLOCK_N];

    float reg_a[TILE];
    float reg_b[TILE];
    float reg_c[TILE][TILE] = {0};

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) << 2;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) << 2;

    int gmem_a_m = smem_a_m + blockIdx.y * BLOCK_M;
    int gmem_b_n = smem_b_n + blockIdx.x * BLOCK_N;

    for(int k = 0; k < K / BLOCK_K; k++){
        int gmem_a_k = smem_a_k + k * BLOCK_K;
        int gmem_b_k = smem_b_k + k * BLOCK_K;

        // Copy
        FLOAT4(shared_a[smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        FLOAT4(shared_b[smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
        __syncthreads();

        // Compute
        for(int kk = 0; kk < BLOCK_K; kk++){
            int ty = threadIdx.y * TILE;
            int tx = threadIdx.x * NUM;

            for(int i = 0; i < TILE; i++) 
                reg_a[i] = shared_a[ty + i][kk];
            
            for(int per = 0; per < 2; per++)
                for(int i = 0; i < NUM; i++)
                    reg_b[i + per * NUM] = shared_b[kk][tx + i + per * BLOCK_N / 2];
            
            // Compute NUM = 4
            for(int i = 0; i < TILE; i++)
                for(int j = 0; j < TILE; j++)
                    reg_c[i][j] += reg_a[i] * reg_b[j];
        }
        __syncthreads();
    }

    // Write Back
    int row = blockIdx.y * BLOCK_M + threadIdx.y * TILE;
    int col = blockIdx.x * BLOCK_N + threadIdx.x * NUM;


    for(int m = 0; m < TILE; m++){
        int r_c_1 = row + m;
        int c_c_1 = col;
        int c_c_2 = c_c_1 + BLOCK_N / 2;
        FLOAT4(c[OFFSET(r_c_1, c_c_1, N)]) = FLOAT4(reg_c[m][0]);
        FLOAT4(c[OFFSET(r_c_1, c_c_2, N)]) = FLOAT4(reg_c[m][NUM]);
    }
}

float sgemm_v5(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;


    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 128, BLOCK_N = 128, TILE = 8;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N / TILE, BLOCK_M / TILE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v5_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecond, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return msecond;
}

// Pipeline
__global__ void sgemm_v6_kernel(float *a, float *b, float*c, int M, int N, int K){
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 8;
    const int TILE = 8, NUM = 4;

    __shared__ float shared_a[2][BLOCK_M][BLOCK_K];
    __shared__ float shared_b[2][BLOCK_K][BLOCK_N];

    float reg_a[TILE];
    float reg_b[TILE];
    float reg_c[TILE][TILE] = {0};

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) << 2;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) << 2;

    int gmem_a_m = smem_a_m + blockIdx.y * BLOCK_M;
    int gmem_b_n = smem_b_n + blockIdx.x * BLOCK_N;

    int gmem_a_k = smem_a_k;
    int gmem_b_k = smem_b_k;

    // Fisrt Stage
    FLOAT4(shared_a[0][smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
    FLOAT4(shared_b[0][smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
    __syncthreads();

    int idx = 0;
    for(int k = 1; k < K / BLOCK_K; k++){
        int gmem_a_k = smem_a_k + k * BLOCK_K;
        int gmem_b_k = smem_b_k + k * BLOCK_K;

        idx = k % 2;
        // Copy
        FLOAT4(shared_a[idx][smem_a_m][smem_a_k]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        FLOAT4(shared_b[idx][smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);

        // Compute
        for(int kk = 0; kk < BLOCK_K; kk++){
            int ty = threadIdx.y * TILE;
            int tx = threadIdx.x * NUM;

            for(int i = 0; i < TILE; i++) 
                reg_a[i] = shared_a[(idx + 1) % 2][ty + i][kk];
            
            for(int per = 0; per < 2; per++)
                for(int i = 0; i < NUM; i++)
                    reg_b[i + per * NUM] = shared_b[(idx + 1) % 2][kk][tx + i + per * BLOCK_N / 2];
            
            // Compute NUM = 4
            for(int i = 0; i < TILE; i++)
                for(int j = 0; j < TILE; j++)
                    reg_c[i][j] += reg_a[i] * reg_b[j];
        }
        __syncthreads();
    }
    // Compute
    for(int kk = 0; kk < BLOCK_K; kk++){
        int ty = threadIdx.y * TILE;
        int tx = threadIdx.x * NUM;

        for(int i = 0; i < TILE; i++) 
            reg_a[i] = shared_a[(idx + 1) % 2][ty + i][kk];
        
        for(int per = 0; per < 2; per++)
            for(int i = 0; i < NUM; i++)
                reg_b[i + per * NUM] = shared_b[(idx + 1) % 2][kk][tx + i + per * BLOCK_N / 2];
        
        // Compute NUM = 4
        for(int i = 0; i < TILE; i++)
            for(int j = 0; j < TILE; j++)
                reg_c[i][j] += reg_a[i] * reg_b[j];
    }
    __syncthreads();


    // Write Back
    int row = blockIdx.y * BLOCK_M + threadIdx.y * TILE;
    int col = blockIdx.x * BLOCK_N + threadIdx.x * NUM;


    for(int m = 0; m < TILE; m++){
        int r_c_1 = row + m;
        int c_c_1 = col;
        int c_c_2 = c_c_1 + BLOCK_N / 2;
        FLOAT4(c[OFFSET(r_c_1, c_c_1, N)]) = FLOAT4(reg_c[m][0]);
        FLOAT4(c[OFFSET(r_c_1, c_c_2, N)]) = FLOAT4(reg_c[m][NUM]);
    }
}

float sgemm_v6(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;


    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 128, BLOCK_N = 128, TILE = 8;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N / TILE, BLOCK_M / TILE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v6_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecond, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return msecond;
}


// Transpose Load A & Pipeline
__global__ void sgemm_v7_kernel(float *a, float *b, float*c, int M, int N, int K){
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 8;
    const int TILE = 8, NUM = 4;

    __shared__ float shared_a[3][BLOCK_K][BLOCK_M];
    __shared__ float shared_b[3][BLOCK_K][BLOCK_N];

    float reg_a[TILE];
    float reg_b[TILE];
    float reg_c[TILE][TILE] = {0};

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) << 2;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) << 2;

    int gmem_a_m = smem_a_m + blockIdx.y * BLOCK_M;
    int gmem_b_n = smem_b_n + blockIdx.x * BLOCK_N;

    int gmem_a_k = smem_a_k;
    int gmem_b_k = smem_b_k;

    // Fisrt Stage
    float load_a[4];

    FLOAT4(load_a[0]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
    shared_a[0][smem_a_k + 0][smem_a_m] = load_a[0];
    shared_a[0][smem_a_k + 1][smem_a_m] = load_a[1];
    shared_a[0][smem_a_k + 2][smem_a_m] = load_a[2];
    shared_a[0][smem_a_k + 3][smem_a_m] = load_a[3];
    FLOAT4(shared_b[0][smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
    __syncthreads();

    int idx = 0;
    for(int k = 1; k < K / BLOCK_K; k++){
        int gmem_a_k = smem_a_k + k * BLOCK_K;
        int gmem_b_k = smem_b_k + k * BLOCK_K;

        idx = k % 2;
        // Copy
        FLOAT4(load_a[0]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        shared_a[idx][smem_a_k + 0][smem_a_m] = load_a[0];
        shared_a[idx][smem_a_k + 1][smem_a_m] = load_a[1];
        shared_a[idx][smem_a_k + 2][smem_a_m] = load_a[2];
        shared_a[idx][smem_a_k + 3][smem_a_m] = load_a[3];
        FLOAT4(shared_b[idx][smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);
        

        // Compute
        for(int kk = 0; kk < BLOCK_K; kk++){
            int ty = threadIdx.y * TILE;
            int tx = threadIdx.x * NUM;

            for(int i = 0; i < TILE; i++) 
                reg_a[i] = shared_a[(idx + 1) % 2][kk][ty + i];
            
            for(int per = 0; per < 2; per++)
                for(int i = 0; i < NUM; i++)
                    reg_b[i + per * NUM] = shared_b[(idx + 1) % 2][kk][tx + i + per * BLOCK_N / 2];
                
            
            // Compute NUM = 4
            for(int i = 0; i < TILE; i++)
                for(int j = 0; j < TILE; j++)
                    reg_c[i][j] += reg_a[i] * reg_b[j];
        }
        __syncthreads();
    }
    // Compute
    for(int kk = 0; kk < BLOCK_K; kk++){
        int ty = threadIdx.y * TILE;
        int tx = threadIdx.x * NUM;

        for(int i = 0; i < TILE; i++) 
            reg_a[i] = shared_a[(idx + 1) % 2][kk][ty + i];
        
        for(int per = 0; per < 2; per++)
            for(int i = 0; i < NUM; i++)
                reg_b[i + per * NUM] = shared_b[(idx + 1) % 2][kk][tx + i + per * BLOCK_N / 2];
            
        // Compute NUM = 4
        for(int i = 0; i < TILE; i++)
            for(int j = 0; j < TILE; j++)
                reg_c[i][j] += reg_a[i] * reg_b[j];
    }
    __syncthreads();





    // Write Back
    int row = blockIdx.y * BLOCK_M + threadIdx.y * TILE;
    int col = blockIdx.x * BLOCK_N + threadIdx.x * NUM;

    for(int m = 0; m < TILE; m++){
        int r_c_1 = row + m;
        int c_c_1 = col;
        int c_c_2 = c_c_1 + BLOCK_N / 2;
        FLOAT4(c[OFFSET(r_c_1, c_c_1, N)]) = FLOAT4(reg_c[m][0]);
        FLOAT4(c[OFFSET(r_c_1, c_c_2, N)]) = FLOAT4(reg_c[m][NUM]);
    }
}

float sgemm_v7(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;


    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 128, BLOCK_N = 128, TILE = 8;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N / TILE, BLOCK_M / TILE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v7_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&msecond, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // for(int i = 0; i < M; i++){
    //     for(int j = 0; j < N; j++)
    //         std::cout << c[OFFSET(i, j, N)] << " ";
    //     std::cout << std::endl;
    // }

    return msecond;
}



// Transose LoadA
__global__ void sgemm_v8_kernel(float *a, float *b, float*c, int M, int N, int K){
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 8;
    const int TILE = 8, NUM = 4;

    __shared__ float shared_a[BLOCK_K][BLOCK_N];
    __shared__ float shared_b[BLOCK_K][BLOCK_N];

    float reg_a[TILE];
    float reg_b[TILE];
    float reg_c[TILE][TILE] = {0};

    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    int smem_a_m = tid / 2;
    int smem_a_k = (tid % 2) << 2;

    int smem_b_k = tid / 32;
    int smem_b_n = (tid % 32) << 2;

    int gmem_a_m = smem_a_m + blockIdx.y * BLOCK_M;
    int gmem_b_n = smem_b_n + blockIdx.x * BLOCK_N;

    float load_a[4];

    for(int k = 0; k < K / BLOCK_K; k++){
        int gmem_a_k = smem_a_k + k * BLOCK_K;
        int gmem_b_k = smem_b_k + k * BLOCK_K;

        // Copy
        FLOAT4(load_a[0]) = FLOAT4(a[OFFSET(gmem_a_m, gmem_a_k, K)]);
        shared_a[smem_a_k + 0][smem_a_m] = load_a[0];
        shared_a[smem_a_k + 1][smem_a_m] = load_a[1];
        shared_a[smem_a_k + 2][smem_a_m] = load_a[2];
        shared_a[smem_a_k + 3][smem_a_m] = load_a[3];
        FLOAT4(shared_b[smem_b_k][smem_b_n]) = FLOAT4(b[OFFSET(gmem_b_k, gmem_b_n, N)]);

        __syncthreads();

        // Compute
        for(int kk = 0; kk < BLOCK_K; kk++){
            int ty = threadIdx.y * TILE;
            int tx = threadIdx.x * NUM;

            for(int i = 0; i < TILE; i++) 
                reg_a[i] = shared_a[kk][ty + i];
            
            for(int per = 0; per < 2; per++)
                for(int i = 0; i < NUM; i++)
                    reg_b[i + per * NUM] = shared_b[kk][tx + i + per * BLOCK_N / 2];
            
            // Compute NUM = 4
            for(int i = 0; i < TILE; i++)
                for(int j = 0; j < TILE; j++)
                    reg_c[i][j] += reg_a[i] * reg_b[j];
        }
        __syncthreads();
    }

    // Write Back
    int row = blockIdx.y * BLOCK_M + threadIdx.y * TILE;
    int col = blockIdx.x * BLOCK_N + threadIdx.x * NUM;


    for(int m = 0; m < TILE; m++){
        int r_c_1 = row + m;
        int c_c_1 = col;
        int c_c_2 = c_c_1 + BLOCK_N / 2;
        FLOAT4(c[OFFSET(r_c_1, c_c_1, N)]) = FLOAT4(reg_c[m][0]);
        FLOAT4(c[OFFSET(r_c_1, c_c_2, N)]) = FLOAT4(reg_c[m][NUM]);
    }
}

float sgemm_v8(float *a, float *b, float *c, int M, int N, int K){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;


    float *da, *db, *dc;
    hipMalloc(&da, size_a);
    hipMalloc(&db, size_b);
    hipMalloc(&dc, size_c);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

    const int BLOCK_M = 128, BLOCK_N = 128, TILE = 8;
    dim3 grid(N / BLOCK_N, M / BLOCK_M);
    dim3 block(BLOCK_N / TILE, BLOCK_M / TILE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msecond = 0.0;

    hipEventRecord(start, 0);
    sgemm_v8_kernel<<<grid, block>>>(da, db, dc, M, N, K);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);


    hipEventElapsedTime(&msecond, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return msecond;
}



// WMMA
__global__ void sgemm_v9(float *a, float *b, float*c, int M, int N, int K){

}

// float sgemm_cublas(float *a, float *b, float *c, int M, int N, int K){

//     hipblasHandle_t handle;
//     CUBLAS_CHECK(hipblasCreate(&handle));

//     float cublas_alpha = 1.0;
//     float cublas_beta = 0;

//     size_t size_a = sizeof(float) * M * K;
//     size_t size_b = sizeof(float) * K * N;
//     size_t size_c = sizeof(float) * M * N;

//     float *da, *db, *dc;
//     CUDA_CHECK(hipMalloc(&da, size_a));
//     CUDA_CHECK(hipMalloc(&db, size_b));
//     CUDA_CHECK(hipMalloc(&dc, size_c));

//     hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
//     hipMemcpy(db, b, size_b, hipMemcpyHostToDevice);

//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);

//     float msecond = 0.0;
//     hipEventRecord(start);
    
//     hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &cublas_alpha, db, N, da, K, &cublas_beta, dc, N);

//     hipEventRecord(stop);
//     hipEventSynchronize(stop);


//     hipEventElapsedTime(&msecond, start, stop);
    
//     hipMemcpy(c, dc, size_c, hipMemcpyDeviceToHost);

//     hipFree(da);
//     hipFree(db);
//     hipFree(dc);
    
//     hipEventDestroy(start);
//     hipEventDestroy(stop);

//     hipblasDestroy(handle);

//     return msecond;
// }

using Func = std::function<float(float *, float *, float *c, int, int, int)>;

void data_init(float *data, const int num) {
    std::uniform_real_distribution<float> float_gen(-1.0f, 1.0f);
    std::default_random_engine rand_engine(time(nullptr));
    for (int i = 0; i < num; i++) {
        data[i] = float_gen(rand_engine);
    }
}

void testPerformance(std::pair<Func, string> func, int M, int N, int K, int nums){
    size_t size_a = sizeof(float) * M * K;
    size_t size_b = sizeof(float) * K * N;
    size_t size_c = sizeof(float) * M * N;

    float *a = (float*)malloc(size_a);
    float *b = (float*)malloc(size_b);
    float *c = (float*)malloc(size_c);

    data_init(a, M * K);
    data_init(b, K * N);
    // for(int i = 0; i < M * K; i++) a[i] = 1.0;
    // for(int i = 0; i < K * N; i++) b[i] = 1.0;

    float avg = 0;
    for(int i = 0; i < nums; i++)
        avg += func.first(a, b, c, M, N, K) / nums;
    
    float tflops = 2.0f * M * N * K / (avg / 1e3) / 1e12;
    
    free(a);
    free(b);
    free(c);

    printf("times: %.6f tflops: %.6f %s\n", avg, tflops, func.second.c_str());
}





int main(){
    int M = 1024, N = 1024, K = 1024;


    testPerformance(std::make_pair(sgemm_v1, "sgemm_v1 - Naive"), M, N, K, 100);
    testPerformance(std::make_pair(sgemm_v2, "sgemm_v2 - Block Tiling"), M, N, K, 100);
    testPerformance(std::make_pair(sgemm_v3, "sgemm_v3 - Thread Tiling"), M, N, K, 100);
    testPerformance(std::make_pair(sgemm_v4, "sgemm_v4 - Warp Tiling"), M, N, K ,100);
    testPerformance(std::make_pair(sgemm_v5, "sgemm_v5 - Bank Free"), M, N, K, 100);
    testPerformance(std::make_pair(sgemm_v6, "sgemm_v6 - Pipeline"), M, N, K, 100);
    testPerformance(std::make_pair(sgemm_v7, "sgemm_v7 - Transpose Load A && Pipeline"), M, N, K, 100);
    testPerformance(std::make_pair(sgemm_v8, "sgemm_v8 - Transpose Load A"), M, N, K, 100);

    // testPerformance(std::make_pair(sgemm_cublas, "sgemm_cublas"), M, N, K, 100);


    return 0;
}